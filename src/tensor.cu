
// #include "tensor.h"

#include <hip/hip_runtime.h>
#include <iostream>
#include <map>

void add(float* a, float* b, float* c, int size){
    for (int i = 0; i < size; i++) {
        c[i] = a[i] + b[i];
    }
}


class tensor{
public:
    tensor(float* data, int* size, int dim, bool autograd, int id = -1){
        this->data = data;
        this->size = size;
        this->dim = dim;
        this->volume = 1;
        this->autograd = autograd;
        for (int i = 0; i < dim; i++) {
            this->volume *= size[i];
        }
        if (id == -1) {
            this->id = 1 + rand() % 100000;
        }
        this->id = id;
    }

    tensor(float* data, int* size, int dim, std::string creation_op, tensor* right, tensor* left, bool autograd){
        this->data = data;
        this->size = size;
        this->dim = dim;
        this->volume = 1;
        for (int i = 0; i < dim; i++) {
            this->volume *= size[i];
        }
        this->right = right;
        this->left = left;
        this->creation_op = creation_op;
        this->autograd = autograd;
        if (id == -1) {
            id = 1 + rand() % 100000;
        }

        if (left != NULL && right != NULL) {
            auto temp = left->children.find(this->id);
            if (temp == left->children.end()) {
                left->children.insert(std::make_pair(id, 1));
            }
            else{
                left->children[id] += 1;
            }

            temp = right->children.find(this->id);
            if (temp == right->children.end()) {
                right->children.insert(std::make_pair(this->id, 1));
            }
            else{
                right->children[this->id] += 1;
            }
        }
    }

    float operator[](int i){
        return data[i];
    }

    bool all_children_grads_accounted_for(){
        auto temp = this->children.begin();
        while (temp != children.end()) {
            if (temp->second != 0) {
                return false;
            }
            temp++;
        }
        return true;
    }

    int* getSize(){
        return size;
    }

    int getVolume(){
        return volume;
    }

    int getDim(){
        return dim;
    }

    void print(){
        if (dim == 2) {
            for (int i = 0; i < size[0]; i++) {
                std::cout << "[";
                for (int j = 0; j < size[1]; j++) {
                    std::cout << std::fixed;
                    std::cout.precision(2);
                    std::cout << data[i * size[1] + j];
                    if (j != size[1] -1 ) {
                        std::cout << ", ";
                    }
                }
                std::cout << "]\n";
            }
        }
        else{
            std::cout << "[";
            for (int i = 0; i < volume; i++) {
                std::cout << data[i];
                if (i != volume -1) {
                    std::cout << ", ";
                }
            }
            std::cout << "]\n";
        }
    }

    void backward(float* grad, tensor* grad_origin = NULL){
        if (autograd) {
            if (grad_origin != NULL) {
                // std::cout << children[grad_origin->id] << '\n';
                if (children[grad_origin->id] == 0) {
                    std::cout << "cannot backprop more than once" << '\n';
                    return;
                }
                else{
                    children[grad_origin->id] -= 1;
                }
            }
            if (this->grad == NULL) {
                this->grad = grad;
            }
            else{
                add(this->grad, grad, this->grad, volume);
            }
            // if (grad_origin != NULL) {
            //     std::cout << children[grad_origin->id] << '\n';
            //     std::cout << all_children_grads_accounted_for() << '\n';
            // }
            if (left != NULL && right != NULL && (all_children_grads_accounted_for() || grad_origin == NULL)  ) {
                if (creation_op == "add") {
                    // std::cout << "2/* message */" << '\n';
                    left->backward(grad, this);
                    right->backward(grad, this);
                }
            }
        }
    }

// private:
    float* grad = NULL;
    bool autograd = false;
    float* data = NULL;
    int* size = NULL;
    int id = -1;
    int dim;
    int volume;
    std::string creation_op;
    tensor* left = NULL;
    tensor* right = NULL;
    std::map<int, int> children;
};

tensor* add (tensor* f, tensor*  t){
    if (f->getVolume() != t->getVolume() && f->getSize()[0] != t->getSize()[0]) {
        // throw "Can not take square root of negative number";
        throw std::invalid_argument("Dim +.");
    }

    float* aret = new float[t->getVolume()];

    for (int i = 0; i < f->getVolume(); i++) {
        aret[i] = (*t)[i] + (*f)[i];
    }
    if (f->autograd && t->autograd) {
        tensor* tret = new tensor(aret, f->getSize(), t->getDim(), "add", t, f, true);
        // std::cout << f->id << '\n';
        return tret;
    }
    tensor* tret = new tensor(aret, f->getSize(), t->getDim(), "add", t, f, false);
    return tret;
}

void print(float* array, int size){
    if (array == NULL) {
        std::cout << "error" << '\n';
        return;
    }
    std::cout << "(";
    for (int i = 0; i < size; i++) {
        std::cout << array[i];
        if (size != i + 1) {
            std::cout << ", ";
        }
    }
    std::cout << ")\n";
}

int main(int argc, char const *argv[]) {
    int size[] = {2,3};
    // int size2[] = {2,4};
    float arr[] =  {1,1,1,1,1,1};
    float arra[] = {2,2,2,3,3,3};
    float arrb[] = {2,2,2,3,3,3};
    float arre[] = {2,2,2,3,3,3};
    // float arrd[] = {4,5,6,7,8,9};
    int dim = 2;
    tensor* a = new tensor(arra, size, dim, true, 1);
    tensor* b = new tensor(arrb, size, dim, true, 2);
    tensor* c = new tensor(arre, size, dim, true, 3);
    // tensor* d = new tensor(arrd, size, dim);

    tensor *d = add(a, b);
    tensor *e = add(b, c);
    tensor *f = add(d, e);
    // f->print();
    f->backward(arr);
    // std::cout << "2123" << '\n';
    print(b->grad, 6);

    // std::map<int, int> test;
    // test.insert(std::make_pair(1, 1));
    // test[1] += 1;
    // std::cout << test[2] << '\n';

    return 0;
}







//
