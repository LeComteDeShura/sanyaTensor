#include "hip/hip_runtime.h"
#include "linearAlg.h"


void matrixMultiplyHost(float * A, float * B, float * C, int numARows, int numAColumns, int numBRows, int numBColumns, int numCRows, int numCColumns){
    int threadsPerBlockDim = TILE_WIDTH;
    dim3 blockDim(threadsPerBlockDim, threadsPerBlockDim, 1); // Grid: X x Y x Z=1

    int blocksPerGridDimX = ceilf(numCColumns / (float)threadsPerBlockDim);
    int blocksPerGridDimY = ceilf(numCRows / (float)threadsPerBlockDim);
    dim3 gridDim(blocksPerGridDimX, blocksPerGridDimY, 1);

    matrixMultiplyKernel<<<gridDim, blockDim>>>(A, B, C, numARows, numAColumns, numBRows, numBColumns, numCRows, numCColumns);

    hipDeviceSynchronize();
}

int matrixMultiply(float * A, float * B, float * C, int numARows, int numAColumns, int numBRows, int numBColumns, int numCRows, int numCColumns){
    float* d_a;
    float* d_b;
    float* d_c;

    hipMalloc((void **)&d_a, numARows * numAColumns*sizeof(float));
    hipMalloc((void **)&d_b, numBRows * numBColumns*sizeof(float));
    hipMalloc((void **)&d_c, numCRows * numCColumns*sizeof(float));

    hipError_t err = hipSuccess;
    err = hipMemcpy(d_a, A, numARows * numAColumns*sizeof(float), hipMemcpyHostToDevice);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device vector A (error code %s)!\n", hipGetErrorString(err));
        return -1;
    }

    err = hipMemcpy(d_b, B, numBRows * numBColumns*sizeof(float), hipMemcpyHostToDevice);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device vector B (error code %s)!\n", hipGetErrorString(err));
        return -1;
    }

    matrixMultiplyHost(d_a, d_b, d_c, numARows, numAColumns, numBRows, numBColumns, numCRows, numCColumns);

    err = hipMemcpy(C, d_c , numARows * numBColumns*sizeof(float), hipMemcpyDeviceToHost);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device vector C (error code %s)!\n", hipGetErrorString(err));
        return -1;
    }

    return 0;
}
__global__ void matrixMultiplyKernel(float * A, float * B, float * C, int numARows, int numAColumns, int numBRows, int numBColumns, int numCRows, int numCColumns) {
    //@@ Insert code to implement matrix multiplication here
    __shared__ float ds_M[TILE_WIDTH][TILE_WIDTH];
    __shared__ float ds_N[TILE_WIDTH][TILE_WIDTH];
    int bx = blockIdx.x, by = blockIdx.y,
       tx = threadIdx.x, ty = threadIdx.y,
       Row = by * TILE_WIDTH + ty,
       Col = bx * TILE_WIDTH + tx;
    float Pvalue = 0;

    for (int m = 0; m < (numAColumns-1)/TILE_WIDTH+1; ++m) {
       if (Row < numARows && m*TILE_WIDTH+tx < numAColumns)
          ds_M[ty][tx] = A[Row*numAColumns + m*TILE_WIDTH+tx];
       else
          ds_M[ty][tx] = 0;
       if (Col < numBColumns && m*TILE_WIDTH+ty < numBRows)
          ds_N[ty][tx] = B[(m*TILE_WIDTH+ty)*numBColumns+Col];
       else
          ds_N[ty][tx] = 0;

       __syncthreads();
       for (int k = 0; k < TILE_WIDTH; ++k)
          Pvalue += ds_M[ty][k] * ds_N[k][tx];
       __syncthreads();
    }
    if (Row < numCRows && Col < numCColumns)
       C[Row*numCColumns+Col] = Pvalue;
}

void matrixMultiply(int * A, int * B, int * C, int numARows, int numAColumns, int numBRows, int numBColumns, int numCRows, int numCColumns){
    int threadsPerBlockDim = TILE_WIDTH;
    dim3 blockDim(threadsPerBlockDim, threadsPerBlockDim, 1); // Grid: X x Y x Z=1

    int blocksPerGridDimX = ceilf(numCColumns / (float)threadsPerBlockDim);
    int blocksPerGridDimY = ceilf(numCRows / (float)threadsPerBlockDim);
    dim3 gridDim(blocksPerGridDimX, blocksPerGridDimY, 1);

    matrixMultiplyKernel<<<gridDim, blockDim>>>(A, B, C, numARows, numAColumns, numBRows, numBColumns, numCRows, numCColumns);
    // printf("\n%d\n", C[0]);
    hipDeviceSynchronize();
}
__global__ void matrixMultiplyKernel(int * A, int * B, int * C, int numARows, int numAColumns, int numBRows, int numBColumns, int numCRows, int numCColumns) {
    //@@ Insert code to implement matrix multiplication here
    __shared__ float ds_M[TILE_WIDTH][TILE_WIDTH];
    __shared__ float ds_N[TILE_WIDTH][TILE_WIDTH];
    int bx = blockIdx.x, by = blockIdx.y,
       tx = threadIdx.x, ty = threadIdx.y,
       Row = by * TILE_WIDTH + ty,
       Col = bx * TILE_WIDTH + tx;
    int Pvalue = 0;

    for (int m = 0; m < (numAColumns-1)/TILE_WIDTH+1; ++m) {
       if (Row < numARows && m*TILE_WIDTH+tx < numAColumns)
          ds_M[ty][tx] = A[Row*numAColumns + m*TILE_WIDTH+tx];
       else
          ds_M[ty][tx] = 0;
       if (Col < numBColumns && m*TILE_WIDTH+ty < numBRows)
          ds_N[ty][tx] = B[(m*TILE_WIDTH+ty)*numBColumns+Col];
       else
          ds_N[ty][tx] = 0;

       __syncthreads();
       for (int k = 0; k < TILE_WIDTH; ++k)
          Pvalue += ds_M[ty][k] * ds_N[k][tx];
       __syncthreads();
    }
    if (Row < numCRows && Col < numCColumns)
       C[Row*numCColumns+Col] = Pvalue;
}

//////
void sigmoida(float *A, float *B, int rows, int cols){
    int threadsPerBlockDim = TILE_WIDTH;
    int blocksPerGridDim = ceilf(rows*cols / (float)threadsPerBlockDim);
    sigmoidaKernel<<<blocksPerGridDim, threadsPerBlockDim>>>(A, B, rows, cols);

    hipDeviceSynchronize();
}
__global__ void sigmoidaKernel(float *A, float *B, int rows, int cols){
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    float e = 2.7180;
    for (int i = index; i < rows*cols; i += stride)
        B[i] = 1 / (1 + pow(e, (A[i] * -1)));
}


///////
void transpose(int *A, int *B, int rows, int cols){
    int threadsPerBlockDim = TILE_WIDTH;
    dim3 blockDim(threadsPerBlockDim, threadsPerBlockDim, 1); // Grid: X x Y x Z=1

    int blocksPerGridDimX = ceilf(cols / (float)threadsPerBlockDim);
    int blocksPerGridDimY = ceilf(rows / (float)threadsPerBlockDim);
    dim3 gridDim(blocksPerGridDimX, blocksPerGridDimY, 1);
    transposeMatrixFast<<<gridDim, blockDim>>>(A, B, cols, rows);

    hipDeviceSynchronize();
}
void transposeHost(float *A, float *B, int rows, int cols){
    int threadsPerBlockDim = TILE_WIDTH;
    dim3 blockDim(threadsPerBlockDim, threadsPerBlockDim, 1); // Grid: X x Y x Z=1

    int blocksPerGridDimX = ceilf(cols / (float)threadsPerBlockDim);
    int blocksPerGridDimY = ceilf(rows / (float)threadsPerBlockDim);
    dim3 gridDim(blocksPerGridDimX, blocksPerGridDimY, 1);
    transposeMatrixFast<<<gridDim, blockDim>>>(A, B, cols, rows);

    hipDeviceSynchronize();
}

int transpose(float *A, float *B, int rows, int cols){
    float* d_a;
    float* d_b;

    hipMalloc((void **)&d_a, rows * cols*sizeof(float));
    hipMalloc((void **)&d_b, rows * cols*sizeof(float));

    hipError_t err = hipSuccess;
    err = hipMemcpy(d_a, A, rows * cols*sizeof(float), hipMemcpyHostToDevice);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device vector A (error code %s)!\n", hipGetErrorString(err));
        return -1;
    }

    transposeHost(d_a, d_b, rows, cols);

    err = hipMemcpy(B, d_b , rows * cols*sizeof(float), hipMemcpyDeviceToHost);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device vector B (error code %s)!\n", hipGetErrorString(err));
        return -1;
    }

    return 0;
}

__global__ void transposeKernel(float *idata, float *odata, int height, int width){
	__shared__ float tile[TILE_DIM][TILE_DIM];
	int xIndex = blockIdx.x * TILE_DIM + threadIdx.x;
	int yIndex = blockIdx.y * TILE_DIM + threadIdx.y;
	int index_in = xIndex + (yIndex)*width;
	xIndex = blockIdx.y * TILE_DIM + threadIdx.x;
	yIndex = blockIdx.x * TILE_DIM + threadIdx.y;
	int index_out = xIndex + (yIndex)*height;
	tile[threadIdx.y][threadIdx.x] = idata[index_in];
	__syncthreads();
	odata[index_out] = tile[threadIdx.x][threadIdx.y];
}
__global__ void transposeMatrixFast(int * inputMatrix, int * outputMatrix, int w, int h){
    __shared__ float temp[BLOCK_SIZE][BLOCK_SIZE];
    int xIndex = blockIdx.x * blockDim.x + threadIdx.x;
    int yIndex = blockIdx.y * blockDim.y + threadIdx.y;
    if ((xIndex < w) && (yIndex < h)){
        int idx = yIndex * w + xIndex;
        temp[threadIdx.y][threadIdx.x] = inputMatrix[idx];
    }
    __syncthreads();
    xIndex = blockIdx.y * blockDim.y + threadIdx.x;
    yIndex = blockIdx.x * blockDim.x + threadIdx.y;
    if ((xIndex < h) && (yIndex < w)){
        int idx = yIndex * h + xIndex;
        outputMatrix[idx] = temp[threadIdx.x][threadIdx.y];
    }
}
__global__ void transposeMatrixFast(float * inputMatrix, float * outputMatrix, int w, int h){
    __shared__ float temp[BLOCK_SIZE][BLOCK_SIZE];
    int xIndex = blockIdx.x * blockDim.x + threadIdx.x;
    int yIndex = blockIdx.y * blockDim.y + threadIdx.y;
    if ((xIndex < w) && (yIndex < h)){
        int idx = yIndex * w + xIndex;
        temp[threadIdx.y][threadIdx.x] = inputMatrix[idx];
    }
    __syncthreads();
    xIndex = blockIdx.y * blockDim.y + threadIdx.x;
    yIndex = blockIdx.x * blockDim.x + threadIdx.y;
    if ((xIndex < h) && (yIndex < w)){
        int idx = yIndex * h + xIndex;
        outputMatrix[idx] = temp[threadIdx.x][threadIdx.y];
    }
}

//////
void subMat(float *A, float *B, float*C, int rows, int cols){
    int threadsPerBlockDim = TILE_DIM;
    int blocksPerGridDim = ceilf(rows*cols / (float)threadsPerBlockDim);
    subKernelMat<<<blocksPerGridDim, threadsPerBlockDim>>>(A, B, C, rows, cols);

    hipDeviceSynchronize();
}
__global__ void subKernelMat(float *A, float *B, float *C, int rows, int cols){
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    for (int i = index; i < rows*cols; i += stride)
        C[i] = A[i] - B[i];
}

/////
void addMat(float *A, float *B, float*C, int rows, int cols){
    int threadsPerBlockDim = TILE_DIM;
    int blocksPerGridDim = ceilf(rows*cols / (float)threadsPerBlockDim);
    addKernelMat<<<blocksPerGridDim, threadsPerBlockDim>>>(A, B, C, rows, cols);

    hipDeviceSynchronize();
}
__global__ void addKernelMat(float *A, float *B, float *C, int rows, int cols){
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;


    for (int i = index; i < rows*cols; i += stride)
        C[i] = A[i] + B[i];
}

////
void mulMat(float *A, float *B, float*C, int rows, int cols){
    int threadsPerBlockDim = TILE_DIM;
    int blocksPerGridDim = ceilf(rows*cols / (float)threadsPerBlockDim);
    mulKernelMat<<<blocksPerGridDim, threadsPerBlockDim>>>(A, B, C, rows, cols);

    hipDeviceSynchronize();
}
__global__ void mulKernelMat(float *A, float *B, float *C, int rows, int cols){
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;


    for (int i = index; i < rows*cols; i += stride)
        C[i] = A[i] * B[i];
}


///
void subNum(float *A, float num, float*C, int rows, int cols){
    int threadsPerBlockDim = TILE_DIM;
    int blocksPerGridDim = ceilf(rows*cols / (float)threadsPerBlockDim);
    subKernelNum<<<blocksPerGridDim, threadsPerBlockDim>>>(A, num, C, rows, cols);

    hipDeviceSynchronize();
}
__global__ void subKernelNum(float *A, float num, float *C, int rows, int cols){
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    for (int i = index; i < rows*cols; i += stride)
        C[i] = num - A[i];
}

///
void addNum(float *A, float num, float*C, int rows, int cols){
    int threadsPerBlockDim = TILE_DIM;
    int blocksPerGridDim = ceilf(rows*cols / (float)threadsPerBlockDim);
    addKernelNum<<<blocksPerGridDim, threadsPerBlockDim>>>(A, num, C, rows, cols);

    hipDeviceSynchronize();
}
__global__ void addKernelNum(float *A, float num, float *C, int rows, int cols){
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    for (int i = index; i < rows*cols; i += stride)
        C[i] = num + A[i];
}


///
void mulNum(float *A, float num, float*C, int rows, int cols){
    int threadsPerBlockDim = TILE_DIM;
    int blocksPerGridDim = ceilf(rows*cols / (float)threadsPerBlockDim);
    mulKernelNum<<<blocksPerGridDim, threadsPerBlockDim>>>(A, num, C, rows, cols);

    hipDeviceSynchronize();
}
__global__ void mulKernelNum(float *A, float num, float *C, int rows, int cols){
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    for (int i = index; i < rows*cols; i += stride)
        C[i] = num * A[i];
}























//
